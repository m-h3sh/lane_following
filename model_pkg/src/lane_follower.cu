#include "hip/hip_runtime.h"
#include "rclcpp/rclcpp.hpp"
#include "sensor_msgs/msg/image.hpp"
#include "geometry_msgs/msg/point.hpp"
#include "bits/stdc++.h"
#include "opencv2/opencv.hpp"
#include "cv_bridge/cv_bridge.h"
// #include <mlpack/methods/dbscan/dbscan.hpp>
#include <mlpack/core.hpp>
#include "mlpack.hpp"
#include <armadillo>
#include <vector>
#include <pcl/point_types.h>
#include <pcl_conversions/pcl_conversions.h>
#include <pcl/io/pcd_io.h>
#include <chrono>
#include <functional>
#include <memory>
#include <string>
#include <Eigen/Dense>

#include "std_msgs/msg/string.hpp"
#include "std_msgs/msg/header.hpp"
#include "sensor_msgs/msg/camera_info.hpp"
#include "sensor_msgs/msg/point_cloud2.hpp"
#include "sensor_msgs/msg/point_field.hpp"
#include <math.h>
#include "geometry_msgs/msg/twist.hpp"
// #include "nav_msgs/msg/odometry.hpp"
#include "dbscan_cuda.cuh"
#include "std_msgs/msg/float32_multi_array.hpp"
// #include "geometry_msgs/msg/pose_stamped.hpp"
// #include "nav2_util/lifecycle_service_client.hpp"
// #include "nav2_msgs/action/navigate_to_pose.hpp"
// #include "nav2_util/simple_action_server.hpp"
// #include "tf2_geometry_msgs/tf2_geometry_msgs.hpp"



using namespace std;
using namespace std::chrono_literals;
using namespace std;
using namespace Eigen;
using std::placeholders::_1;
using namespace cv;



class LaneFollower : public rclcpp::Node
{
public:
    LaneFollower() : Node("lane_follower")
    {   
        this->declare_parameter("bt_low",180);
	    this->declare_parameter("bt_high",255);
        subscription = this->create_subscription<sensor_msgs::msg::Image>(
            "/zed/masked_image", 10, std::bind(&LaneFollower::binary_thresholding, this, std::placeholders::_1));
        
        bt_low = this->get_parameter("bt_low").as_int();
        bt_high = this->get_parameter("bt_high").as_int();
        subscription_caminfo = this->create_subscription<sensor_msgs::msg::CameraInfo>(
            "/zed/zed_node/rgb/camera_info", 10,
            std::bind(&LaneFollower::call, this, _1));
        publisher_far = this->create_publisher<std_msgs::msg::Float32MultiArray>("/far_ipm", 10);
        publisher_near = this->create_publisher<sensor_msgs::msg::PointCloud2>("/near_ipm", 10);

        db_publisher = this->create_publisher<sensor_msgs::msg::Image>("/dbImage", 10);
        thresh_publisher = this->create_publisher<sensor_msgs::msg::Image>("/threshImage", 10);
    }

private:

    void binary_thresholding(const sensor_msgs::msg::Image::SharedPtr msg)
    {   
        ////cout<<" RECEIVED IMAGE"<<endl; 
        cv_bridge::CvImagePtr cv_ptr = cv_bridge::toCvCopy(msg, sensor_msgs::image_encodings::BGR8);
        cv::Mat cv_image = cv_ptr->image, gray_image, thresholded_image;
        int rows = cv_image.rows;
        int cols = cv_image.cols;
        // cv_image = cv_image(cv::Range(rows/2,rows),cv::Range::all());
        cv::medianBlur(cv_image,cv_image,5);
        cv::cvtColor(cv_image, gray_image,CV_BGR2GRAY); 
        // cv::imshow("window", cv_image);
        // cv::waitKey(0);
        for (int y=0 ; y< rows/4 ; y++) {
            for (int x=0 ; x< cols ; x++) {
                gray_image.at<uchar>(y, x) = 0;
            }
        }
        ////cout<<"BINARY THRESHOLDING STARTED"<<endl;
        // cv::inRange(cv_image, Scalar(110, 0, 0), Scalar(145, 255, 255), thresholded_image);
        cv::inRange(gray_image, bt_low, bt_high, thresholded_image);
        threshMsg = cv_bridge::CvImage(std_msgs::msg::Header(),"mono8",thresholded_image).toImageMsg();
        thresh_publisher->publish(*threshMsg);
        ////cout<<"BINARY THRESHOLDING DONE"<<endl;

        
        std::vector<cv::Point> white_pixel_indices;

        // cv::findNonZero(thresholded_image, white_pixel_indices);
        

        // ////cout<<"COLLECTING WHITE POINTS ENDED"<<endl;

        // arma::mat data(2, white_pixel_indices.size());
        
        // for (size_t i = 0; i < white_pixel_indices.size(); ++i) {
        //     data(0, i) = white_pixel_indices[i].x;
        //     data(1, i) = white_pixel_indices[i].y;
        // }

        // data.resize(2, (white_pixel_indices.size()+1)/2);

        // ////cout<<white_pixel_indices.size()<<endl;
        // ////cout<<"SETTING PARAMETERS FOR DB SCAN"<<endl;
        // // Parameters for DBSCAN
        // double epsilon = 5.0; // Adjust as needed
        // size_t minPoints = 10; // Adjust as needed
        int eps = 10;
        // //cout << "nigga2" << endl;
        
        // cv::Canny(thresholded_image,thresholded_image,50,75, 3,true);

        // Mat dilated;
        // float dilation_size = 0.5;  // Adjust the dilation size as needed
        // Mat element = getStructuringElement(MORPH_RECT, Size(2 * dilation_size + 1, 2 * dilation_size + 1), cv::Point(dilation_size, dilation_size));
        // dilate(thresholded_image, thresholded_image, element);
        
        
        
        
        // cv::imshow("window",thresholded_image);
        // cv::waitKey(10);

        Graph graph(thresholded_image, eps);
        DBSCAN scanner(&graph, 0.1f, 10);
        scanner.identify_cluster();

        // //cout<<"nigga"<<endl;
        arma::Row<size_t> assignments(graph.size());

        for (int i =0 ; i < graph.size() ; i++) {
            // //cout<<"nigga3"<<endl;
            assignments[i] = scanner.label(i);
            // //cout << assignments[i]<<" ";
        }
        // //cout<<endl;F

        for (int i =0 ; i<graph.size() ; i++) {
            cv::Point point;
            point.x = graph.node(i).y;
            point.y = graph.node(i).x;
            
            white_pixel_indices.push_back(point);
            
        }

        // ////cout<<"DBSCAN STARTING"<<endl;
        // mlpack::dbscan::DBSCAN<> dbscan(epsilon, minPoints);
        // dbscan.Cluster(data, assignments);
        // ////cout<<"DBSCAN ENDED"<<endl;
        std::unordered_map<size_t, size_t> clusterSizes;

  // Count points in each cluster
        ////cout <<"Count points in each cluster" <<endl;
        
        for (size_t i = 0; i < assignments.n_elem; ++i) {
            if (assignments[i] != SIZE_MAX) { // Ignore noise points
            clusterSizes[assignments[i]]++;
            }
        }

        ////cout<<"Create a vector of pairs (clusterID, size) and sort it"<<endl;
        std::vector<std::pair<size_t, size_t>> sortedClusters(clusterSizes.begin(), clusterSizes.end());
        
        std::sort(sortedClusters.begin(), sortedClusters.end(), [](const std::pair<size_t, size_t>& a, const std::pair<size_t, size_t>& b) {
            return b.second < a.second; // Sort in descending order of size
        });
        
        

        if (sortedClusters.size() < 2) {
            // RCLCPP_WARN(this->get_logger(), "Not enough clusters found");
            return;
        }


        if (!sortedClusters.empty()) {
            if (sortedClusters[0].second == 0) {
                ////cout<<"No Clusters Found" <<endl;
                return;
            }
            largestClusterID = sortedClusters[0].first;
            SecondLargestClusterID = sortedClusters[1].first;


        }
        else {
            ////cout<<"No sorted clusters"<<endl;
            return;
        }
        
        vector<cv::Point> largestCluster, secondLargestCluster;
        ////cout<<"test"<<endl;
        for (size_t i = 0; i<assignments.n_elem ; ++i) {
            if (assignments[i] == largestClusterID) largestCluster.push_back(white_pixel_indices[i]);
            else if (assignments[i] == SecondLargestClusterID) secondLargestCluster.push_back(white_pixel_indices[i]);
        }
        
        
        
        
        cv::Mat dbImage = cv::Mat::zeros(gray_image.rows, gray_image.cols,CV_8UC1);
        cv::Mat dbImage2 = cv::Mat::zeros(gray_image.rows, gray_image.cols,CV_8UC1);

        

        for (cv::Point point : largestCluster) {
            dbImage.at<uchar>(point.y,point.x) = 255;
            dbImage2.at<uchar>(point.y,point.x) = 255;
            
        }

        for (cv::Point point : secondLargestCluster) {
            dbImage2.at<uchar>(point.y,point.x) = 255;
        }

       
        dbMsg = cv_bridge::CvImage(std_msgs::msg::Header(),"mono8",dbImage2).toImageMsg();
        db_publisher->publish(*dbMsg);
       
        midpoint_publisher(largestCluster,secondLargestCluster,dbImage2);
    

    }
    


    void midpoint_publisher(vector<cv::Point> largestCluster, vector<cv::Point> secondLargestCluster, cv::Mat lanes_binary) {

        geometry_msgs::msg::Point left,right,mid_farthest, mid_closest;
        int y__cord, x1__cord, x2__cord;

        bool midpoint_flag_farthest = false;
        bool midpoint_flag_closest = false;

        int current_y=0, prev_y=0;

        

        // Finding farthest point
        for (int j = 0 ; j<secondLargestCluster.size() ; j++) {
            current_y = secondLargestCluster[j].y;
            if (current_y != 0 && prev_y !=0 && current_y == prev_y) continue;  
            for (int i =0 ; i<largestCluster.size(); ++i) {
                if (largestCluster[i].y > secondLargestCluster[j].y) break;
                if ((largestCluster[i].y <= secondLargestCluster[j].y + 5)&&(largestCluster[i].y >= secondLargestCluster[j].y - 5)) {
                        if (!largestCluster[i].x or !secondLargestCluster[j].x)
                        mid_farthest.y = largestCluster[i].y;
                        mid_farthest.x = (largestCluster[i].x + secondLargestCluster[j].x)/2;
                        midpoint_flag_farthest = true;
                        break;
                } 
            }
            if (midpoint_flag_farthest) break;
            prev_y = current_y;
        }

        // Finding Closest point

        current_y = 0;
        prev_y = 0;

        // std::sort(secondLargestCluster.end()-50, secondLargestCluster.end(), 
        //       [](const cv::Point& a, const cv::Point& b) {
        //           return a.y < b.y;
        //       });
     
        
        for (int j =secondLargestCluster.size() ; j>0 ; j--) {

            // //cout <<secondLargestCluster[j] <<" "<< largestCluster[j]<<endl;

            current_y = secondLargestCluster[j].y;
            if (current_y != 0 && prev_y !=0 && current_y == prev_y) continue;  
            if ((current_y - mid_farthest.y)<10) continue;
            if (current_y ==0 ) continue;
            for (int i =largestCluster.size() ; i>0 ; i--) {
                if ((largestCluster[i].y <= secondLargestCluster[j].y + 5)&&(largestCluster[i].y >= secondLargestCluster[j].y - 5)) {

                        
                        y__cord = largestCluster[i].y;
                        x1__cord = largestCluster[i].x;
                        x2__cord = secondLargestCluster[j].x;
                      
                        // //cout<<" Y_coord "<<y__cord<<" "<<j<<endl;
                        
                        if (abs(x1__cord - x2__cord)>50 && y__cord != 0) {
                            mid_closest.y = y__cord;
                            mid_closest.x = (x1__cord + x2__cord)/2;
                            midpoint_flag_closest = true;
                            break;
                        }
                } 
            }
            if (midpoint_flag_closest) break;
            prev_y = current_y;
        }

        // vector<geometry_msgs::msg::Point> mid_far_vect(10);
        // count++;
        
        
        ////cout<<midpoint_flag_farthest<<endl;
        
        if (midpoint_flag_farthest) {
            // Add the new midpoint to the deque
            smoothed_midpoints_far.push_back(mid_farthest);
            if (smoothed_midpoints_far.size() > smoothing_window_size) {
                smoothed_midpoints_far.pop_front();
            }

            // Calculate the average midpoint
            geometry_msgs::msg::Point avg_farthest = calculate_average_point(smoothed_midpoints_far);
            lanes_binary.at<uchar>(avg_farthest.y, avg_farthest.x) = 255;
            if (cam_info_received) {
                std_msgs::msg::Float32MultiArray cloud_far = process_point(avg_farthest.y, avg_farthest.x);
                cam_info_received = false;
                publisher_far->publish(cloud_far);
            }
            
        }

        if (midpoint_flag_closest) {
            // Add the new midpoint to the deque
            smoothed_midpoints_near.push_back(mid_closest);
            if (smoothed_midpoints_near.size() > smoothing_window_size) {
                smoothed_midpoints_near.pop_front();
            }

            // Calculate the average midpoint
            geometry_msgs::msg::Point avg_closest = calculate_average_point(smoothed_midpoints_near);
            lanes_binary.at<uchar>(avg_closest.y, avg_closest.x) = 255;
            if (cam_info_received) {
                // cout<<avg_closest.y<<" "<<avg_closest.x<<endl;
                // sensor_msgs::msg::PointCloud2 cloud_near = process_point(avg_closest.y, avg_closest.x);
                cam_info_received = false;
                // publisher_near->publish(cloud_near);
            }
            
            
        }

        
        

        // cv::imshow("window",lanes_binary);
        // cv::waitKey(30);
        // process_point(y__cord,(x1__cord+x2__cord)/2);


        
        
        // publish_vect(mid_farthest,mid_closest);




        
        // cv:imshow("window",lanes_binary);
        // cv::waitKey(30);

        


    }
    

    void call(const sensor_msgs::msg::CameraInfo::SharedPtr msg) {
        this->camera_info = *msg;
        cam_info_received = true;
    }
    // void call_odom(const nav_msgs::msg::Odometry::SharedPtr msg) {
    //     this->odom = *msg;
    //     odom_received = true;

    // }


 std_msgs::msg::Float32MultiArray process_point(int y, int x) {
    
    sensor_msgs::msg::PointCloud2 pub_pointcloud;
    auto cloud_msg = std::make_unique<PointCloud>();

    // Process the single point

    float roll = 0;
    float pitch = -24 * M_PI / 180;
    float yaw = 0;
    float h = 1.41;

    vector<double> k(9);

    double cy, cr, sy, sr, sp, cp;
    cy = cos(yaw);
    sy = sin(yaw);
    cp = cos(pitch);
    sp = sin(pitch);
    cr = cos(roll);
    sr = sin(roll);
    
    k[0] = cr * cy + sp * sr + sy;
    k[1] = cr * sp * sy - cy * sr;
    k[2] = -cp * sy;
    k[3] = cp * sr;
    k[4] = cp * cr;
    k[5] = sp;
    k[6] = cr * sy - cy * sp * sr;
    k[7] = -cr * cy * sp - sr * sy;
    k[8] = cp * cy;

    Eigen::Matrix3d K;
    K << k[0], k[1], k[2],
         k[3], k[4], k[5],
         k[6], k[7], k[8];
    
    Eigen::Vector3d nor(0.0, 1.0, 0.0);
    
           
    
// Calculate UV
    Eigen::Vector3d nc = K*nor;


    // Inverse camera matrix
    
    auto caminfo = this->camera_info.k;
    
    Eigen::Map<Eigen::Matrix3d> kin(caminfo.data());
    kin = kin.inverse();
    

  
    Eigen::Vector3d uv_hom = { x, y, 1 };


    
    Eigen::Vector3d kin_uv = kin*uv_hom;
    double denom = kin_uv.dot(nc);


    std::vector<float> vec(3);
    vec[0] = h * kin_uv[2] / denom;
    vec[1] = -h * kin_uv[0] / denom;
    vec[2] = 0;
    //----------------------------------------------------------
    // cloud_msg->points.push_back(vec);

    // Prepare the PointCloud message for publishing
    // cloud_msg->height = 1;
    // cloud_msg->width = cloud_msg->points.size();
    // cloud_msg->is_dense = false;
    // pcl::toROSMsg(*cloud_msg, pub_pointcloud);
    // pub_pointcloud.header.frame_id = "base_link";
    // pub_pointcloud.header.stamp = rclcpp::Clock().now();


    // // Return the cloud
    // for (int i =0 ; i< pub_pointcloud.data.size() ; i++) {
    //     cout<<pub_pointcloud.data[i]<<" ";
    // }
    // cout<<endl;
    // return pub_pointcloud;
//--------------------------------------------------------
    pub_array.set__data(vec);
    return pub_array;
    
    }
    geometry_msgs::msg::Point calculate_average_point(const std::deque<geometry_msgs::msg::Point>& points) {
        geometry_msgs::msg::Point avg_point;
        avg_point.x = 0;
        avg_point.y = 0;
        avg_point.z = 0;
        for (const auto& point : points) {
            avg_point.x += point.x;
            avg_point.y += point.y;
            avg_point.z += point.z;
        }
        avg_point.x /= points.size();
        avg_point.y /= points.size();
        avg_point.z /= points.size();
        return avg_point;
    }
 

// Start of Defining Variables ------------------------------------------------------------
    int largestClusterID=0, SecondLargestClusterID=0;
    rclcpp::Subscription<sensor_msgs::msg::Image>::SharedPtr subscription;
    int bt_low, bt_high;
    // rclcpp::Subscription<sensor_msgs::msg::Image>::SharedPtr subscription2;
    // // rclcpp::Publisher<geometry_msgs::msg::Point>::SharedPtr l_publisher;
    // // rclcpp::Publisher<geometry_msgs::msg::Point>::SharedPtr r_publisher;
    // rclcpp::Publisher<geometry_msgs::msg::Point>::SharedPtr m_publisher;

    // rclcpp::Publisher<geometry_msgs::msg::Point>::SharedPtr ma_publisher;
    // rclcpp::Publisher<geometry_msgs::msg::Point>::SharedPtr ma2_publisher;
    rclcpp::Publisher<sensor_msgs::msg::Image>::SharedPtr db_publisher;
    rclcpp::Publisher<sensor_msgs::msg::Image>::SharedPtr thresh_publisher;

    sensor_msgs::msg::Image::SharedPtr dbMsg;
    sensor_msgs::msg::Image::SharedPtr threshMsg;

    rclcpp::Publisher<geometry_msgs::msg::Point>::SharedPtr m_publisher;
    rclcpp::Publisher<geometry_msgs::msg::Point>::SharedPtr vecto;
    rclcpp::Publisher<geometry_msgs::msg::Twist>::SharedPtr cmd_vel_pub;

    rclcpp::Subscription<sensor_msgs::msg::CameraInfo>::SharedPtr subscription_caminfo;
    // rclcpp::Subscription<nav_msgs::msg::Odometry>::SharedPtr subscription_odom;

    rclcpp::Publisher<std_msgs::msg::Float32MultiArray>::SharedPtr publisher_far;
    rclcpp::Publisher<sensor_msgs::msg::PointCloud2>::SharedPtr publisher_near;

    sensor_msgs::msg::CameraInfo camera_info;
    size_t smoothing_window_size = 10;
    std::deque<geometry_msgs::msg::Point> smoothed_midpoints_far;
    std::deque<geometry_msgs::msg::Point> smoothed_midpoints_near;

    std_msgs::msg::Float32MultiArray pub_array;
    // nav_msgs::msg::Odometry odom;
    bool odom_received = false;
    typedef pcl::PointCloud<pcl::PointXYZ> PointCloud;
    int count =0;
    bool cam_info_received = false;

// End of Defining Variables

};




int main(int argc, char * argv[]) {
    rclcpp::init(argc, argv);
    auto lane_follower = std::make_shared<LaneFollower>();
    
    // Example: process a single point (x, y)
    // ipm_node->process_point(100, 150); // Replace (100, 150) with desired point coordinates
    
    rclcpp::spin(lane_follower);
    rclcpp::shutdown();
    return 0;
}

